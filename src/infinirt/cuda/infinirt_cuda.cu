#include "../../utils.h"
#include "infinirt_cuda.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <map>
#include <memory>
#include <string.h>
#include <variant>

#define CHECK_CUDART(RT_API) CHECK_INTERNAL(RT_API, hipSuccess)

namespace infinirt::cuda {
// Internal struct definitions for opaque pointers
struct PhyMemImpl {
    infinirtAllocationHandle_t handle;
    size_t len;
    hipMemAllocationProp *prop;
};

using VacantRegion = size_t;
using MappedRegion = std::shared_ptr<PhyMemImpl>;
using PhyRegion = std::variant<MappedRegion, VacantRegion>;

struct VirtualMemManagerImpl {
    infinirtDeviceptr_t device_ptr;
    size_t len;
    std::map<size_t, PhyRegion> map;
};

infiniStatus_t getDeviceCount(int *count) {
    CHECK_CUDART(hipGetDeviceCount(count));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t setDevice(int device_id) {
    CHECK_CUDART(hipSetDevice(device_id));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t deviceSynchronize() {
    CHECK_CUDART(hipDeviceSynchronize());
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamCreate(infinirtStream_t *stream_ptr) {
    hipStream_t stream;
    CHECK_CUDART(hipStreamCreate(&stream));
    *stream_ptr = stream;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamDestroy(infinirtStream_t stream) {
    CHECK_CUDART(hipStreamDestroy((hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamSynchronize(infinirtStream_t stream) {
    CHECK_CUDART(hipStreamSynchronize((hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamWaitEvent(infinirtStream_t stream, infinirtEvent_t event) {
#ifdef ENABLE_ILUVATAR_API
    return INFINI_STATUS_NOT_IMPLEMENTED;
#else
    CHECK_CUDART(hipStreamWaitEvent((hipStream_t)stream, (hipEvent_t)event));
    return INFINI_STATUS_SUCCESS;
#endif
}

infiniStatus_t eventCreate(infinirtEvent_t *event_ptr) {
    hipEvent_t event;
    CHECK_CUDART(hipEventCreate(&event));
    *event_ptr = event;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventRecord(infinirtEvent_t event, infinirtStream_t stream) {
    CHECK_CUDART(hipEventRecord((hipEvent_t)event, (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventQuery(infinirtEvent_t event, infinirtEventStatus_t *status_ptr) {
    auto status = hipEventQuery((hipEvent_t)event);
    if (status == hipSuccess) {
        *status_ptr = INFINIRT_EVENT_COMPLETE;
    } else if (status == hipErrorNotReady) {
        *status_ptr = INFINIRT_EVENT_NOT_READY;
    } else {
        CHECK_CUDART(status);
    }
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventSynchronize(infinirtEvent_t event) {
    CHECK_CUDART(hipEventSynchronize((hipEvent_t)event));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventDestroy(infinirtEvent_t event) {
    CHECK_CUDART(hipEventDestroy((hipEvent_t)event));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mallocDevice(void **p_ptr, size_t size) {
    CHECK_CUDART(hipMalloc(p_ptr, size));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mallocHost(void **p_ptr, size_t size) {
    CHECK_CUDART(hipHostMalloc(p_ptr, size));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t freeDevice(void *ptr) {
    CHECK_CUDART(hipFree(ptr));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t freeHost(void *ptr) {
    CHECK_CUDART(hipHostFree(ptr));
    return INFINI_STATUS_SUCCESS;
}

hipMemcpyKind toCudaMemcpyKind(infinirtMemcpyKind_t kind) {
    switch (kind) {
    case INFINIRT_MEMCPY_H2D:
        return hipMemcpyHostToDevice;
    case INFINIRT_MEMCPY_D2H:
        return hipMemcpyDeviceToHost;
    case INFINIRT_MEMCPY_D2D:
        return hipMemcpyDeviceToDevice;
    case INFINIRT_MEMCPY_H2H:
        return hipMemcpyHostToHost;
    default:
        return hipMemcpyDefault;
    }
}

infiniStatus_t memcpy(void *dst, const void *src, size_t size, infinirtMemcpyKind_t kind) {
    CHECK_CUDART(hipMemcpy(dst, src, size, toCudaMemcpyKind(kind)));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t memcpyAsync(void *dst, const void *src, size_t size, infinirtMemcpyKind_t kind, infinirtStream_t stream) {
    CHECK_CUDART(hipMemcpyAsync(dst, src, size, toCudaMemcpyKind(kind), (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mallocAsync(void **p_ptr, size_t size, infinirtStream_t stream) {
    CHECK_CUDART(hipMallocAsync(p_ptr, size, (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t freeAsync(void *ptr, infinirtStream_t stream) {
    CHECK_CUDART(hipFreeAsync(ptr, (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

hipMemAllocationProp *getMemProp() {
    int device_id;
    infinirtGetDevice(nullptr, &device_id);
    hipMemAllocationProp *cuda_prop = new hipMemAllocationProp();
    memset(cuda_prop, 0, sizeof(hipMemAllocationProp));
    cuda_prop->type = hipMemAllocationTypePinned;
    cuda_prop->requestedHandleTypes = hipMemHandleTypeNone;
    cuda_prop->location.type = hipMemLocationTypeDevice;
    cuda_prop->location.id = device_id;
    return cuda_prop;
}

infiniStatus_t getMemGranularityMinimum(size_t *granularity) {
    hipMemAllocationProp *cuda_prop = getMemProp();
    CHECK_CUDART(hipMemGetAllocationGranularity(granularity, cuda_prop, hipMemAllocationGranularityMinimum));

    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t createPhysicalMem(infinirtPhyMem_t *phy_mem, size_t len) {
    hipMemGenericAllocationHandle_t handle;
    hipMemAllocationProp *cuda_prop = getMemProp();
    CHECK_CUDART(hipMemCreate(&handle, len, cuda_prop, 0));

    PhyMemImpl *impl = new PhyMemImpl;
    impl->handle = (infinirtAllocationHandle_t)handle;
    impl->len = len;
    impl->prop = cuda_prop;

    *phy_mem = (infinirtPhyMem_t)impl;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t releasePhysicalMem(infinirtPhyMem_t phy_mem) {
    PhyMemImpl *impl = (PhyMemImpl *)phy_mem;
    CHECK_CUDART(hipMemRelease((hipMemGenericAllocationHandle_t)impl->handle));
    delete impl->prop;
    delete impl;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t createVirtualMem(infinirtVirtualMem_t *vm, size_t len) {
    hipDeviceptr_t device_ptr;
    CHECK_CUDART(hipMemAddressReserve(&device_ptr, len, 0, (hipDeviceptr_t)0, 0));

    VirtualMemManagerImpl *impl = new VirtualMemManagerImpl;
    impl->device_ptr = (infinirtDeviceptr_t)device_ptr;
    impl->len = len;
    impl->map.clear();
    impl->map[0] = VacantRegion(len);

    *vm = (infinirtVirtualMem_t)impl;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t releaseVirtualMem(infinirtVirtualMem_t vm) {
    VirtualMemManagerImpl *impl = (VirtualMemManagerImpl *)vm;

    // First unmap all mapped regions
    for (auto &[offset, region] : impl->map) {
        if (auto *mapped = std::get_if<MappedRegion>(&region)) {
            hipDeviceptr_t ptr = (hipDeviceptr_t)impl->device_ptr + offset;
            CHECK_CUDART(hipMemUnmap(ptr, (*mapped)->len));
        }
    }

    // Then free the virtual address space
    CHECK_CUDART(hipMemAddressFree((hipDeviceptr_t)impl->device_ptr, impl->len));

    delete impl;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mapVirtualMem(void **mapped_ptr, infinirtVirtualMem_t vm, size_t offset,
                             infinirtPhyMem_t phy_mem) {
    VirtualMemManagerImpl *vm_impl = (VirtualMemManagerImpl *)vm;
    PhyMemImpl *phy_impl = (PhyMemImpl *)phy_mem;

    if (offset > vm_impl->len || offset + phy_impl->len > vm_impl->len) {
        std::cerr << "Offset is out of range"
                  << " offset: " << offset << " phy_mem->len: " << phy_impl->len << " vm->len: " << vm_impl->len << std::endl;
        return INFINI_STATUS_BAD_PARAM;
    }
    auto it = vm_impl->map.upper_bound(offset);
    --it;
    auto &[head, region] = *it;

    if (auto *vacant = std::get_if<VacantRegion>(&region)) {
        if (phy_impl->len > *vacant) {
            std::cerr << "Physical memory length is greater than the vacant region length" << std::endl;
            return INFINI_STATUS_BAD_PARAM;
        }

        hipDeviceptr_t ptr = (hipDeviceptr_t)vm_impl->device_ptr + offset;
        CHECK_CUDART(hipMemMap(ptr, phy_impl->len, 0, (hipMemGenericAllocationHandle_t)phy_impl->handle, 0));
        hipMemAccessDesc desc = {};
        desc.location = phy_impl->prop->location;
        desc.flags = hipMemAccessFlagsProtReadWrite;
        CHECK_CUDART(hipMemSetAccess(ptr, phy_impl->len, &desc, 1));

        vm_impl->map.erase(it);
        vm_impl->map[offset] = std::make_shared<PhyMemImpl>(*phy_impl);
        auto head_len = offset - head;
        auto tail_len = *vacant - head_len - phy_impl->len;
        if (head_len > 0) {
            vm_impl->map[head] = head_len;
        }
        if (tail_len > 0) {
            vm_impl->map[head + head_len + phy_impl->len] = tail_len;
        }

        *mapped_ptr = (void *)ptr;
        return INFINI_STATUS_SUCCESS;
    } else {
        std::cerr << "Virtual memory already mapped at offset: " << offset << std::endl;
        return INFINI_STATUS_INTERNAL_ERROR;
    }
}

infiniStatus_t unmapVirtualMem(infinirtVirtualMem_t vm, size_t offset) {
    VirtualMemManagerImpl *vm_impl = (VirtualMemManagerImpl *)vm;
    auto it = vm_impl->map.find(offset);
    if (it == vm_impl->map.end()) {
        return INFINI_STATUS_BAD_PARAM;
    }

    if (auto *mapped = std::get_if<MappedRegion>(&it->second)) {
        auto phy_mem = *mapped;
        auto ptr = (hipDeviceptr_t)vm_impl->device_ptr + offset;
        CHECK_CUDART(hipMemUnmap(ptr, phy_mem->len));

        it->second = phy_mem->len;
        return INFINI_STATUS_SUCCESS;
    } else {
        return INFINI_STATUS_BAD_PARAM;
    }
}

} // namespace infinirt::cuda
