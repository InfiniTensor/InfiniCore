#include "../../utils.h"
#include "infinirt_cuda.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string.h>

#define CHECK_CUDART(RT_API) CHECK_INTERNAL(RT_API, hipSuccess)

namespace infinirt::cuda {

infiniStatus_t getDeviceCount(int *count) {
    CHECK_CUDART(hipGetDeviceCount(count));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t setDevice(int device_id) {
    CHECK_CUDART(hipSetDevice(device_id));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t deviceSynchronize() {
    CHECK_CUDART(hipDeviceSynchronize());
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamCreate(infinirtStream_t *stream_ptr) {
    hipStream_t stream;
    CHECK_CUDART(hipStreamCreate(&stream));
    *stream_ptr = stream;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamDestroy(infinirtStream_t stream) {
    CHECK_CUDART(hipStreamDestroy((hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamSynchronize(infinirtStream_t stream) {
    CHECK_CUDART(hipStreamSynchronize((hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamWaitEvent(infinirtStream_t stream, infinirtEvent_t event) {
#ifdef ENABLE_ILUVATAR_API
    return INFINI_STATUS_NOT_IMPLEMENTED;
#else
    CHECK_CUDART(hipStreamWaitEvent((hipStream_t)stream, (hipEvent_t)event));
    return INFINI_STATUS_SUCCESS;
#endif
}

infiniStatus_t eventCreate(infinirtEvent_t *event_ptr) {
    hipEvent_t event;
    CHECK_CUDART(hipEventCreate(&event));
    *event_ptr = event;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventRecord(infinirtEvent_t event, infinirtStream_t stream) {
    CHECK_CUDART(hipEventRecord((hipEvent_t)event, (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventQuery(infinirtEvent_t event, infinirtEventStatus_t *status_ptr) {
    auto status = hipEventQuery((hipEvent_t)event);
    if (status == hipSuccess) {
        *status_ptr = INFINIRT_EVENT_COMPLETE;
    } else if (status == hipErrorNotReady) {
        *status_ptr = INFINIRT_EVENT_NOT_READY;
    } else {
        CHECK_CUDART(status);
    }
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventSynchronize(infinirtEvent_t event) {
    CHECK_CUDART(hipEventSynchronize((hipEvent_t)event));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventDestroy(infinirtEvent_t event) {
    CHECK_CUDART(hipEventDestroy((hipEvent_t)event));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mallocDevice(void **p_ptr, size_t size) {
    CHECK_CUDART(hipMalloc(p_ptr, size));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mallocHost(void **p_ptr, size_t size) {
    CHECK_CUDART(hipHostMalloc(p_ptr, size));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t freeDevice(void *ptr) {
    CHECK_CUDART(hipFree(ptr));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t freeHost(void *ptr) {
    CHECK_CUDART(hipHostFree(ptr));
    return INFINI_STATUS_SUCCESS;
}

hipMemcpyKind toCudaMemcpyKind(infinirtMemcpyKind_t kind) {
    switch (kind) {
    case INFINIRT_MEMCPY_H2D:
        return hipMemcpyHostToDevice;
    case INFINIRT_MEMCPY_D2H:
        return hipMemcpyDeviceToHost;
    case INFINIRT_MEMCPY_D2D:
        return hipMemcpyDeviceToDevice;
    case INFINIRT_MEMCPY_H2H:
        return hipMemcpyHostToHost;
    default:
        return hipMemcpyDefault;
    }
}

infiniStatus_t memcpy(void *dst, const void *src, size_t size, infinirtMemcpyKind_t kind) {
    CHECK_CUDART(hipMemcpy(dst, src, size, toCudaMemcpyKind(kind)));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t memcpyAsync(void *dst, const void *src, size_t size, infinirtMemcpyKind_t kind, infinirtStream_t stream) {
    CHECK_CUDART(hipMemcpyAsync(dst, src, size, toCudaMemcpyKind(kind), (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mallocAsync(void **p_ptr, size_t size, infinirtStream_t stream) {
    CHECK_CUDART(hipMallocAsync(p_ptr, size, (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t freeAsync(void *ptr, infinirtStream_t stream) {
    CHECK_CUDART(hipFreeAsync(ptr, (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

hipMemAllocationProp *getMemProp() {
    int device_id;
    infinirtGetDevice(nullptr, &device_id);
    hipMemAllocationProp *cuda_prop = new hipMemAllocationProp();
    memset(cuda_prop, 0, sizeof(hipMemAllocationProp));
    cuda_prop->type = hipMemAllocationTypePinned;
    cuda_prop->requestedHandleTypes = hipMemHandleTypeNone;
    cuda_prop->location.type = hipMemLocationTypeDevice;
    cuda_prop->location.id = device_id;
    return cuda_prop;
}

infiniStatus_t getMemGranularityMinimum(size_t *granularity) {
    hipMemAllocationProp *cuda_prop = getMemProp();
    CHECK_CUDART(hipMemGetAllocationGranularity(granularity, cuda_prop, hipMemAllocationGranularityMinimum));

    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t createPhysicalMem(infinirtPhysicalMemoryHandle_t *pm_handle, size_t len) {
    hipMemGenericAllocationHandle_t handle;
    hipMemAllocationProp *cuda_prop = getMemProp();
    CHECK_CUDART(hipMemCreate(&handle, len, cuda_prop, 0));

    *pm_handle = (infinirtPhysicalMemoryHandle_t)handle;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t releasePhysicalMem(infinirtPhysicalMemoryHandle_t pm_handle) {
    CHECK_CUDART(hipMemRelease((hipMemGenericAllocationHandle_t)pm_handle));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t createVirtualMem(void **vm, size_t len) {
    hipDeviceptr_t device_ptr;
    CHECK_CUDART(hipMemAddressReserve(&device_ptr, len, 0, (hipDeviceptr_t)0, 0));

    *vm = (void *)device_ptr;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t releaseVirtualMem(void *vm, size_t len) {
    CHECK_CUDART(hipMemAddressFree((hipDeviceptr_t)vm, len));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mapVirtualMem(void *vm, size_t len, size_t offset,
                             infinirtPhysicalMemoryHandle_t pm_handle) {

    hipDeviceptr_t ptr = (hipDeviceptr_t)vm + offset;
    CHECK_CUDART(hipMemMap(ptr, len, 0, (hipMemGenericAllocationHandle_t)pm_handle, 0));

    hipMemAllocationProp *cuda_prop = getMemProp();
    hipMemAccessDesc desc = {};
    desc.location = cuda_prop->location;
    desc.flags = hipMemAccessFlagsProtReadWrite;
    CHECK_CUDART(hipMemSetAccess(ptr, len, &desc, 1));

    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t unmapVirtualMem(void *vm, size_t len) {
    hipDeviceptr_t ptr = (hipDeviceptr_t)vm;
    CHECK_CUDART(hipMemUnmap(ptr, len));

    return INFINI_STATUS_SUCCESS;
}

} // namespace infinirt::cuda
