#include "../../utils.h"
#include "infinirt_cuda.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string.h>

#define CHECK_CUDART(RT_API) CHECK_INTERNAL(RT_API, hipSuccess)

namespace infinirt::cuda {
infiniStatus_t getDeviceCount(int *count) {
    CHECK_CUDART(hipGetDeviceCount(count));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t setDevice(int device_id) {
    CHECK_CUDART(hipSetDevice(device_id));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t deviceSynchronize() {
    CHECK_CUDART(hipDeviceSynchronize());
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamCreate(infinirtStream_t *stream_ptr) {
    hipStream_t stream;
    CHECK_CUDART(hipStreamCreate(&stream));
    *stream_ptr = stream;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamDestroy(infinirtStream_t stream) {
    CHECK_CUDART(hipStreamDestroy((hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamSynchronize(infinirtStream_t stream) {
    CHECK_CUDART(hipStreamSynchronize((hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t streamWaitEvent(infinirtStream_t stream, infinirtEvent_t event) {
#ifdef ENABLE_ILUVATAR_API
    return INFINI_STATUS_NOT_IMPLEMENTED;
#else
    CHECK_CUDART(hipStreamWaitEvent((hipStream_t)stream, (hipEvent_t)event));
    return INFINI_STATUS_SUCCESS;
#endif
}

infiniStatus_t eventCreate(infinirtEvent_t *event_ptr) {
    hipEvent_t event;
    CHECK_CUDART(hipEventCreate(&event));
    *event_ptr = event;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventRecord(infinirtEvent_t event, infinirtStream_t stream) {
    CHECK_CUDART(hipEventRecord((hipEvent_t)event, (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventQuery(infinirtEvent_t event, infinirtEventStatus_t *status_ptr) {
    auto status = hipEventQuery((hipEvent_t)event);
    if (status == hipSuccess) {
        *status_ptr = INFINIRT_EVENT_COMPLETE;
    } else if (status == hipErrorNotReady) {
        *status_ptr = INFINIRT_EVENT_NOT_READY;
    } else {
        CHECK_CUDART(status);
    }
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventSynchronize(infinirtEvent_t event) {
    CHECK_CUDART(hipEventSynchronize((hipEvent_t)event));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t eventDestroy(infinirtEvent_t event) {
    CHECK_CUDART(hipEventDestroy((hipEvent_t)event));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mallocDevice(void **p_ptr, size_t size) {
    CHECK_CUDART(hipMalloc(p_ptr, size));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mallocHost(void **p_ptr, size_t size) {
    CHECK_CUDART(hipHostMalloc(p_ptr, size));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t freeDevice(void *ptr) {
    CHECK_CUDART(hipFree(ptr));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t freeHost(void *ptr) {
    CHECK_CUDART(hipHostFree(ptr));
    return INFINI_STATUS_SUCCESS;
}

hipMemcpyKind toCudaMemcpyKind(infinirtMemcpyKind_t kind) {
    switch (kind) {
    case INFINIRT_MEMCPY_H2D:
        return hipMemcpyHostToDevice;
    case INFINIRT_MEMCPY_D2H:
        return hipMemcpyDeviceToHost;
    case INFINIRT_MEMCPY_D2D:
        return hipMemcpyDeviceToDevice;
    case INFINIRT_MEMCPY_H2H:
        return hipMemcpyHostToHost;
    default:
        return hipMemcpyDefault;
    }
}

infiniStatus_t memcpy(void *dst, const void *src, size_t size, infinirtMemcpyKind_t kind) {
    CHECK_CUDART(hipMemcpy(dst, src, size, toCudaMemcpyKind(kind)));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t memcpyAsync(void *dst, const void *src, size_t size, infinirtMemcpyKind_t kind, infinirtStream_t stream) {
    CHECK_CUDART(hipMemcpyAsync(dst, src, size, toCudaMemcpyKind(kind), (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mallocAsync(void **p_ptr, size_t size, infinirtStream_t stream) {
    CHECK_CUDART(hipMallocAsync(p_ptr, size, (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t freeAsync(void *ptr, infinirtStream_t stream) {
    CHECK_CUDART(hipFreeAsync(ptr, (hipStream_t)stream));
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t getMemProp(infinirtMemProp_t *prop_ptr, infiniDevice_t device, int device_id) {
    hipMemAllocationProp *cuda_prop = new hipMemAllocationProp();
    memset(cuda_prop, 0, sizeof(hipMemAllocationProp));
    cuda_prop->type = hipMemAllocationTypePinned;
    cuda_prop->requestedHandleTypes = hipMemHandleTypeNone;
    cuda_prop->location.type = hipMemLocationTypeDevice;
    cuda_prop->location.id = device_id;

    *prop_ptr = cuda_prop;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t getMemGranularityMinimum(size_t *granularity, infinirtMemProp_t prop) {
    CHECK_CUDART(hipMemGetAllocationGranularity(granularity, (hipMemAllocationProp *)prop, hipMemAllocationGranularityMinimum));

    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t createPhysicalMem(infinirtPhyMem *phy_mem, size_t len, infinirtMemProp_t prop) {
    hipMemGenericAllocationHandle_t handle;
    hipMemAllocationProp *cuda_prop = (hipMemAllocationProp *)prop;
    CHECK_CUDART(hipMemCreate(&handle, len, (hipMemAllocationProp *)prop, 0));
    phy_mem->handle = (infinirtAllocationHandle_t)handle;
    phy_mem->len = len;
    phy_mem->prop = prop;
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t createVirtualMemManager(infinirtVirtualMemManager *vm, infiniDevice_t device, size_t len, size_t min_addr) {
    hipDeviceptr_t device_ptr;
    CHECK_CUDART(hipMemAddressReserve(&device_ptr, len, 0, (hipDeviceptr_t)min_addr, 0));
    vm->device_ptr = (infinirtDeviceptr_t)device_ptr;
    vm->len = len;
    vm->map.clear();
    vm->map[0] = infinirtVacantRegion(len);
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t mapVirtualMem(void **mapped_ptr, infinirtVirtualMemManager *vm, size_t offset,
                             infinirtPhyMem *phy_mem) {
    if (offset > vm->len || offset + phy_mem->len > vm->len) {
        std::cerr << "Offset is out of range"
                  << " offset: " << offset << " phy_mem->len: " << phy_mem->len << " vm->len: " << vm->len << std::endl;
        return INFINI_STATUS_BAD_PARAM;
    }
    auto it = vm->map.upper_bound(offset);
    --it;
    auto &[head, region] = *it;

    if (auto *vacant = std::get_if<infinirtVacantRegion>(&region)) {
        if (phy_mem->len > *vacant) {
            std::cerr << "Physical memory length is greater than the vacant region length" << std::endl;
            return INFINI_STATUS_BAD_PARAM;
        }

        hipDeviceptr_t ptr = (hipDeviceptr_t)vm->device_ptr + offset;
        CHECK_CUDART(hipMemMap(ptr, phy_mem->len, 0, (hipMemGenericAllocationHandle_t)phy_mem->handle, 0));
        hipMemAccessDesc desc = {};
        auto prop = (hipMemAllocationProp *)phy_mem->prop;
        desc.location = prop->location;
        desc.flags = hipMemAccessFlagsProtReadWrite;
        CHECK_CUDART(hipMemSetAccess(ptr, phy_mem->len, &desc, 1));

        vm->map.erase(it);
        vm->map[offset] = std::make_shared<infinirtPhyMem>(*phy_mem);
        auto head_len = offset - head;
        auto tail_len = *vacant - head_len - phy_mem->len;
        if (head_len > 0) {
            vm->map[head] = head_len;
        }
        if (tail_len > 0) {
            vm->map[head + head_len + phy_mem->len] = tail_len;
        }

        *mapped_ptr = (void *)ptr;
        return INFINI_STATUS_SUCCESS;
    } else {
        std::cerr << "Virtual memory already mapped at offset: " << offset << std::endl;
        return INFINI_STATUS_INTERNAL_ERROR;
    }
}

infiniStatus_t unmapVirtualMem(infinirtVirtualMemManager *vm, size_t offset) {
    auto it = vm->map.find(offset);
    if (it == vm->map.end()) {
        return INFINI_STATUS_BAD_PARAM;
    }

    if (auto *mapped = std::get_if<infinirtMappedRegion>(&it->second)) {
        auto phy_mem = *mapped;
        auto ptr = (hipDeviceptr_t)vm->device_ptr + offset;
        CHECK_CUDART(hipMemUnmap(ptr, phy_mem->len));

        it->second = phy_mem->len;
        return INFINI_STATUS_SUCCESS;
    } else {
        return INFINI_STATUS_BAD_PARAM;
    }
}

} // namespace infinirt::cuda
