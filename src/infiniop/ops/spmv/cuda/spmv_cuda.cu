#include "../../../devices/cuda/cuda_common.cuh"
#include "../../../devices/cuda/cuda_handle.cuh"
#include "../../../devices/cuda/cuda_kernel_common.cuh"
#include "../info.h"
#include "spmv_cuda.cuh"
#include <cstdint>
#include <hipsparse.h>

namespace op::spmv::cuda {

struct Descriptor::Opaque {
  std::shared_ptr<device::cuda::Handle::Internal> internal;
};

Descriptor::~Descriptor() { delete _opaque; }

infiniStatus_t Descriptor::create(infiniopHandle_t handle_,
                                  Descriptor **desc_ptr, size_t num_cols,
                                  size_t num_rows, size_t nnz,
                                  infiniDtype_t dtype) {

  auto handle = reinterpret_cast<device::cuda::nvidia::Handle *>(handle_);

  // currently only float32 supported
  if (dtype != INFINI_DTYPE_F32) {
    return INFINI_STATUS_BAD_TENSOR_DTYPE;
  }

  auto result = SpMVInfo::create(num_cols, num_rows, nnz);
  CHECK_RESULT(result);

  *desc_ptr =
      new Descriptor(dtype, result.take(), new Opaque{handle->internal()},
                     handle->device, handle->device_id);
  return INFINI_STATUS_SUCCESS;
}

infiniStatus_t Descriptor::calculate(void *y, const void *x, const void *values,
                                     const void *row_ptr,
                                     const void *col_indices,
                                     void *stream) const {

  // do basic validation
  auto validation_result =
      validateSpMVCSR(y, x, values, row_ptr, col_indices, _dtype);
  CHECK_OR_RETURN(validation_result == INFINI_STATUS_SUCCESS,
                  validation_result);

  // set up data types and constants
  hipDataType cuda_dtype = HIP_R_32F;
  const float alpha = 1.0f, beta = 0.0f;

  CHECK_STATUS(_opaque->internal->useCusparse(
      (hipStream_t)stream, [&](hipsparseHandle_t cusparse_handle) {
        // create sparse matrix descriptor
        hipsparseSpMatDescr_t mat_descr;
        CHECK_CUSPARSE(hipsparseCreateCsr(
            &mat_descr, _info.num_rows, _info.num_cols, _info.nnz,
            const_cast<void *>(row_ptr), const_cast<void *>(col_indices),
            const_cast<void *>(values), HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO, cuda_dtype));

        // create dense vector descriptors
        hipsparseDnVecDescr_t vec_x, vec_y;
        CHECK_CUSPARSE(hipsparseCreateDnVec(&vec_x, _info.num_cols,
                                           const_cast<void *>(x), cuda_dtype));
        CHECK_CUSPARSE(
            hipsparseCreateDnVec(&vec_y, _info.num_rows, y, cuda_dtype));

        // compute buffer size
        size_t buffer_size = 0;
        CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
            cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha,
            mat_descr, vec_x, &beta, vec_y, cuda_dtype,
            HIPSPARSE_SPMV_ALG_DEFAULT, &buffer_size));

        // allocate external buffer if needed
        void *external_buffer = nullptr;
        if (buffer_size > 0) {
          CHECK_CUDA(hipMalloc(&external_buffer, buffer_size));
        }

        // perform the sparse matrix-vector multiplication
        auto result =
            hipsparseSpMV(cusparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                         &alpha, mat_descr, vec_x, &beta, vec_y, cuda_dtype,
                         HIPSPARSE_SPMV_ALG_DEFAULT, external_buffer);

        // clean up resources
        if (external_buffer) {
          hipFree(external_buffer);
        }
        hipsparseDestroyDnVec(vec_x);
        hipsparseDestroyDnVec(vec_y);
        hipsparseDestroySpMat(mat_descr);

        CHECK_CUSPARSE(result);
        return INFINI_STATUS_SUCCESS;
      }));

  return INFINI_STATUS_SUCCESS;
}

} // namespace op::spmv::cuda
