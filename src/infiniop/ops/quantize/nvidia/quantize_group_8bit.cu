#include "hip/hip_runtime.h"
#include <hip/hip_fp8.h>

#include "../../../devices/nvidia/nvidia_handle.cuh"
#include "quantize_group_8bit.cuh"
#include "quantize_group_8bit_nvidia.cuh"
#include <cmath>

namespace op::quantize::nvidia {

struct Descriptor::Opaque {
    std::shared_ptr<device::nvidia::Handle::Internal> internal;
};

Descriptor::~Descriptor() { delete _opaque; }

infiniStatus_t Descriptor::create(infiniopHandle_t handle_,
                                  Descriptor **desc_ptr,
                                  infiniopTensorDescriptor_t input_desc,
                                  infiniopTensorDescriptor_t output_q_desc,
                                  infiniopTensorDescriptor_t output_s_desc) {
    auto handle = reinterpret_cast<device::nvidia::Handle *>(handle_);
    auto dtype = output_q_desc->dtype();
    CHECK_DTYPE(dtype, INFINI_DTYPE_I8, INFINI_DTYPE_F8_E4M3, INFINI_DTYPE_F8_UE8M0);
    auto result = QuantizeInfo::create(input_desc, output_q_desc, output_s_desc);

    *desc_ptr = new Descriptor(dtype, result.take(), 0,
                               new Opaque{handle->internal()},
                               handle->device, handle->device_id);
    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t
Descriptor::calculate(void *workspace,
                      size_t workspace_size,
                      void *input,
                      void *output_q,
                      void *output_s,
                      int group_size,
                      double eps,
                      double min_8bit,
                      double max_8bit,
                      bool scale_ue8m0,
                      void *stream) const {
    auto hip_stream = reinterpret_cast<hipStream_t>(stream);
    const int num_groups = _info.input()->numel() / group_size;
    constexpr int THREADS_PER_GROUP = 16;

    int groups_per_block = 1;

    if (num_groups % 16 == 0) {
        groups_per_block = 16;
    } else if (num_groups % 8 == 0) {
        groups_per_block = 8;
    } else if (num_groups % 4 == 0) {
        groups_per_block = 4;
    } else if (num_groups % 2 == 0) {
        groups_per_block = 2;
    }

    auto dst_type = _info.output_q()->dtype();
    const int num_blocks = num_groups / groups_per_block;
    const int num_threads = groups_per_block * THREADS_PER_GROUP;

    const bool is_column_major = _info.output_s()->stride(0) < _info.output_s()->stride(1);
    const int hidden_dim = _info.input()->shape()[_info.input()->ndim() - 1];
    const int num_groups_per_row = hidden_dim / group_size;
    const int scale_stride = _info.output_s()->stride(1);
#define LAUNCH_KERNEL(T, DST_DTYPE)                                                \
    do {                                                                           \
        dim3 grid(num_blocks);                                                     \
        dim3 block(num_threads);                                                   \
        if (is_column_major) {                                                     \
            if (scale_ue8m0) {                                                     \
                per_token_group_quant_8bit_kernel<T, DST_DTYPE, true, true>        \
                    <<<grid, block, 0, hip_stream>>>(                             \
                        static_cast<T *>(input), output_q,                         \
                        static_cast<uint32_t *>(output_s), group_size,             \
                        num_groups, groups_per_block, (float)eps, (float)min_8bit, \
                        (float)max_8bit, num_groups_per_row, scale_stride);        \
            } else {                                                               \
                per_token_group_quant_8bit_kernel<T, DST_DTYPE, true, false>       \
                    <<<grid, block, 0, hip_stream>>>(                             \
                        static_cast<T *>(input), output_q,                         \
                        static_cast<float *>(output_s), group_size,                \
                        num_groups, groups_per_block, (float)eps, (float)min_8bit, \
                        (float)max_8bit, num_groups_per_row, scale_stride);        \
            }                                                                      \
        } else {                                                                   \
            assert(!scale_ue8m0);                                                  \
            per_token_group_quant_8bit_kernel<T, DST_DTYPE, false>                 \
                <<<grid, block, 0, hip_stream>>>(                                 \
                    static_cast<T *>(input), output_q,                             \
                    static_cast<float *>(output_s), group_size,                    \
                    num_groups, groups_per_block, (float)eps, (float)min_8bit,     \
                    (float)max_8bit);                                              \
        }                                                                          \
    } while (0)

    if (_info.input()->dtype() == INFINI_DTYPE_F16 && _dtype == INFINI_DTYPE_F8_E4M3) {
        switch (_dtype) {
        case INFINI_DTYPE_F8_E4M3:
            LAUNCH_KERNEL(half, __hip_fp8_e4m3_fnuz);
            break;
        default:
            return INFINI_STATUS_BAD_TENSOR_DTYPE;
        }
    }
#undef LAUNCH_KERNEL
    return INFINI_STATUS_SUCCESS;
}

} // namespace op::quantize::nvidia