#include "hip/hip_runtime.h"
#include "../../../devices/nvidia/nvidia_common.cuh"
#include "rms_norm_nvidia.cuh"

#include "../../../devices/nvidia/nvidia_kernel_common.cuh"
#include <cub/block/block_reduce.cuh>

#include "../../../reduce/cuda/reduce.cuh"

#include "../cuda/kernel.cuh"

#ifdef ENABLE_NINETOOTHED
#include "../../../../../build/ninetoothed/rms_norm.h"
#include "../../../ninetoothed/utils.h"
#endif

template <unsigned int BLOCK_SIZE, typename Tcompute, typename Tdata, typename Tweight>
INFINIOP_CUDA_KERNEL rmsnormKernel(
    Tdata *__restrict__ y,
    ptrdiff_t stride_y_batch,
    ptrdiff_t stride_y_nhead,
    const Tdata *__restrict__ x,
    ptrdiff_t stride_x_batch,
    ptrdiff_t stride_x_nhead,
    const Tweight *__restrict__ w,
    size_t nhead,
    size_t dim,
    float epsilon) {
    rmsnormBlock<BLOCK_SIZE, Tcompute>(y, stride_y_batch, stride_y_nhead, x, stride_x_batch, stride_x_nhead, w, nhead, dim, epsilon);
}

namespace op::rms_norm::nvidia {

struct Descriptor::Opaque {
    std::shared_ptr<device::nvidia::Handle::Internal> internal;
};

Descriptor::~Descriptor() {
    delete _opaque;
}

infiniStatus_t Descriptor::create(
    infiniopHandle_t handle,
    Descriptor **desc_ptr,
    infiniopTensorDescriptor_t y_desc,
    infiniopTensorDescriptor_t x_desc,
    infiniopTensorDescriptor_t w_desc,
    float epsilon) {
    auto result = RMSNormInfo::create(y_desc, x_desc, w_desc, epsilon);
    CHECK_RESULT(result);
    auto info = result.take();

    *desc_ptr = new Descriptor(
        new Opaque{reinterpret_cast<device::nvidia::Handle *>(handle)->internal()},
        std::move(info),
        0,
        handle->device, handle->device_id);
    return INFINI_STATUS_SUCCESS;
}

// launch kernel with different data types
template <unsigned int BLOCK_SIZE>
infiniStatus_t launchKernel(
    uint32_t batch_size, size_t nhead, size_t dim,
    void *y, infiniDtype_t atype, ptrdiff_t stride_y_batch, ptrdiff_t stride_y_nhead,
    const void *x, ptrdiff_t stride_x_batch, ptrdiff_t stride_x_nhead,
    const void *w, infiniDtype_t wtype,
    float epsilon,
    hipStream_t hip_stream) {

#define LAUNCH_KERNEL(Tdata, Tweight, Tcompute)                                                              \
    rmsnormKernel<BLOCK_SIZE, Tcompute, Tdata, Tweight><<<batch_size * nhead, BLOCK_SIZE, 0, hip_stream>>>( \
        reinterpret_cast<Tdata *>(y),                                                                        \
        stride_y_batch,                                                                                      \
        stride_y_nhead,                                                                                      \
        reinterpret_cast<const Tdata *>(x),                                                                  \
        stride_x_batch,                                                                                      \
        stride_x_nhead,                                                                                      \
        reinterpret_cast<const Tweight *>(w),                                                                \
        nhead,                                                                                               \
        dim,                                                                                                 \
        epsilon)

    if (atype == INFINI_DTYPE_F16 && wtype == INFINI_DTYPE_F16) {
        LAUNCH_KERNEL(half, half, float);
    } else if (atype == INFINI_DTYPE_F16 && wtype == INFINI_DTYPE_F32) {
        LAUNCH_KERNEL(half, float, float);
    } else if (atype == INFINI_DTYPE_BF16 && wtype == INFINI_DTYPE_BF16) {
        LAUNCH_KERNEL(__hip_bfloat16, __hip_bfloat16, float);
    } else if (atype == INFINI_DTYPE_BF16 && wtype == INFINI_DTYPE_F32) {
        LAUNCH_KERNEL(__hip_bfloat16, float, float);
    } else if (atype == INFINI_DTYPE_F32 && wtype == INFINI_DTYPE_F32) {
        LAUNCH_KERNEL(float, float, float);
    } else {
        return INFINI_STATUS_BAD_TENSOR_DTYPE;
    }

#undef LAUNCH_KERNEL

    return INFINI_STATUS_SUCCESS;
}

infiniStatus_t Descriptor::calculate(
    void *workspace, size_t workspace_size,
    void *y, const void *x, const void *w,
    void *stream) const {

    if (workspace_size < _workspace_size) {
        return INFINI_STATUS_INSUFFICIENT_WORKSPACE;
    }

    auto stride_x_batch = _info.x_strides[0];
    auto stride_x_nhead = _info.x_strides[1];
    auto stride_y_batch = _info.y_strides[0];
    auto stride_y_nhead = _info.y_strides[1];
    auto dim = _info.dim();
    uint32_t batch_size = static_cast<uint32_t>(_info.shape[0]);
    size_t nhead = _info.shape.size() > 2 ? _info.shape[1] : 1;
    auto hip_stream = reinterpret_cast<hipStream_t>(stream);

#ifndef ENABLE_NINETOOTHED
    // launch kernel with different block sizes
    if (_opaque->internal->maxThreadsPerBlock() == CUDA_BLOCK_SIZE_1024) {
        CHECK_STATUS(launchKernel<CUDA_BLOCK_SIZE_1024>(batch_size, nhead, dim, y, _info.atype, stride_y_batch, stride_y_nhead, x, stride_x_batch, stride_x_nhead, w, _info.wtype, _info.epsilon, hip_stream));
    } else if (_opaque->internal->maxThreadsPerBlock() == CUDA_BLOCK_SIZE_512) {
        CHECK_STATUS(launchKernel<CUDA_BLOCK_SIZE_512>(batch_size, nhead, dim, y, _info.atype, stride_y_batch, stride_y_nhead, x, stride_x_batch, stride_x_nhead, w, _info.wtype, _info.epsilon, hip_stream));
    } else if (_opaque->internal->maxThreadsPerBlock() == CUDA_BLOCK_SIZE_4096) {
        CHECK_STATUS(launchKernel<CUDA_BLOCK_SIZE_4096>(batch_size, nhead, dim, y, _info.atype, stride_y_batch, stride_y_nhead, x, stride_x_batch, stride_x_nhead, w, _info.wtype, _info.epsilon, hip_stream));
    } else {
        return INFINI_STATUS_DEVICE_ARCHITECTURE_NOT_SUPPORTED;
    }
#else
    const auto &ndim{_info.ndim()};

    auto input{ninetoothed::Tensor{x, _info.shape, _info.x_strides}};
    auto weight{ninetoothed::Tensor{w, {_info.shape[ndim - 1]}, {_info.x_strides[ndim - 1]}}.expand_as(input)};
    auto eps{ninetoothed::Tensor<float>{_info.epsilon}};
    auto output{ninetoothed::Tensor{y, _info.shape, _info.y_strides}};
    auto num_normalized_elements{ninetoothed::Tensor<uint64_t>{dim}};

    constexpr auto num_normalized_dims{1};
    const auto &input_dtype{_info.atype};
    const auto &weight_dtype{_info.wtype};
    const auto &output_dtype{_info.atype};
    constexpr auto block_size{1024};

    if (launch_rms_norm(stream, input, weight, eps, output, num_normalized_elements, ndim, num_normalized_dims, input_dtype, weight_dtype, output_dtype, block_size)) {
        return INFINI_STATUS_INTERNAL_ERROR;
    }
#endif
    return INFINI_STATUS_SUCCESS;
}
} // namespace op::rms_norm::nvidia
